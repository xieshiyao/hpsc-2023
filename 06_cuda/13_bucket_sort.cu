#include <hip/hip_runtime.h>

#include <cstdio>
#include <cstdlib>
#include <vector>

__device__ void scan(int* a, int* b, int N)
{
	int i = threadIdx.x;
	for (int j = 1; j < N; j <<= 1)
	{
		b[i] = a[i];
		__syncthreads();
		if (i >= j)
			a[i] += b[i - j];
		__syncthreads();
	}
}

__global__ void bucket_sort(int* key, int n, int range)
{
	int i = threadIdx.x;
	extern __shared__ int s[];
	int* bucket = s;
	int* offset = &s[n];
	int* temp = &s[2 * n];

	atomicAdd(&bucket[key[i]], 1);
	__syncthreads();
	if (i >= 1)
		offset[i] = bucket[i - 1];
	__syncthreads();
	scan(offset, temp, n);
	if (i < range)
	{
		int j = offset[i];
		for (; bucket[i] > 0; bucket[i]--)
			key[j++] = i;
	}
}

int main()
{
	int n = 50;
	int range = 5;
	int* key;
	hipMallocManaged(&key, n * sizeof(int));
	for (int i = 0; i < n; i++)
	{
		key[i] = rand() % range;
		printf("%d ", key[i]);
	}
	printf("\n");

	bucket_sort<<<1, n, 3 * n * sizeof(int)>>>(key, n, range);
	hipDeviceSynchronize();

	for (int i = 0; i < n; i++)
		printf("%d ", key[i]);
	printf("\n");
	hipFree(key);
}
